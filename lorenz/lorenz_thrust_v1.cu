#include "hip/hip_runtime.h"
/* The Thrust version of the Lorenz ensemble example.
   The state type is thrust::device_vector<double>.
 */
#include <iostream>
#include <iomanip>
#include <cmath>
#include <utility>
#include <cstdlib>

#include <thrust/device_vector.h>
#include <thrust/functional.h>

#include <boost/timer/timer.hpp>
#include <boost/typeof/typeof.hpp>

#ifdef USE_VANILLA_ODEINT
#  include <boost/numeric/odeint.hpp>
#  include <boost/numeric/odeint/external/vexcl/vexcl.hpp>
   namespace odeint = boost::numeric::odeint;
#else
#  include "thrust_operations.hpp"
#  include "thrust_algebra.hpp"
#  include "runge_kutta4.hpp"
   namespace odeint = ncwg;
#endif

#include "log.hpp"

//---------------------------------------------------------------------------
typedef thrust::device_vector<double> state_type;
struct lorenz_system {
    size_t n;
    double sigma, b;
    const state_type &R;

    lorenz_system(size_t n, const state_type &R,
            double sigma = 10.0, double b = 8.0 / 3.0
            ) : n(n), R(R), sigma(sigma), b(b) { }

    struct lorenz_functor {
        double sigma, b;
        lorenz_functor(double sigma, double b)
            : sigma(sigma), b(b) {}

        template<class T>
        __host__ __device__ void operator()(T t) const {
            double x = thrust::get<0>( t );
            double y = thrust::get<1>( t );
            double z = thrust::get<2>( t );
            double R = thrust::get<3>( t );

            thrust::get<4>( t ) = sigma * ( y - x );
            thrust::get<5>( t ) = R * x - y - x * z;
            thrust::get<6>( t ) = -b * z + x * y;
        }
    };

    template<class State, class Deriv>
    void operator()(const State &x, Deriv &dxdt, double t) const {
        BOOST_AUTO(start,
                thrust::make_zip_iterator( thrust::make_tuple(
                        x.begin(),
                        x.begin() + n,
                        x.begin() + 2 * n,
                        R.begin(),
                        dxdt.begin(),
                        dxdt.begin() + n,
                        dxdt.begin() + 2 * n
                        ) )
                );

        thrust::for_each(start, start+n, lorenz_functor(sigma, b));
    }
};

//---------------------------------------------------------------------------
int main(int argc, char *argv[]) {
    const size_t n = argc > 1 ? atoi(argv[1]) : 1024;
    const double dt = 0.01;
    const double t_max = 10.0;

    thrust::host_vector<double> r(n);
    const double Rmin = 0.1, Rmax = 50.0;
    for(size_t i = 0; i < n; ++i)
        r[i] = Rmin + i * (Rmax - Rmin) / (n - 1);

    state_type R = r;
    state_type x(3 * n);

    // initialize x,y,z
    thrust::fill(x.begin(), x.end(), 10.0);

    odeint::runge_kutta4_classic<
        state_type, double, state_type, double,
        odeint::thrust_algebra, odeint::thrust_operations
        > stepper;

    lorenz_system sys(n, R);

#ifndef CPU_RUN
    hipDeviceSynchronize();
#endif
    boost::timer::cpu_timer timer;

    for(double t = 0; t < t_max; t += dt)
        stepper.do_step(sys, x, t, dt);

#ifdef CPU_RUN
    log_perf("cpu_v1", n, t_max / dt, timer.elapsed());
#else
    hipDeviceSynchronize();
    log_perf("thrust_v1", n, t_max / dt, timer.elapsed());
#endif

#ifdef SHOW_OUTPUT
    thrust::host_vector<double> res = x;
    std::cout << "x = {" << std::setprecision(6);
    for(size_t i = 0; i < n; ++i) {
        if (i % 2 == 0) std::cout << "\n" << std::setw(6) << i << ":";
        std::cout << std::scientific << " (";
        for(size_t j = 0; j < 3; ++j)
            std::cout << std::setw(14) << res[j * 3 + i];
        std::cout << ")";
    }
    std::cout << "\n}" << std::endl;
#endif

    return 0;
}
